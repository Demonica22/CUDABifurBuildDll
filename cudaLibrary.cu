#include "hip/hip_runtime.h"
#include "cudaLibrary.cuh"

// ---------------------------------------------------------------------------------
// --- Âû÷èñëÿåò ñëåäóþùåå çíà÷åíèå äèñêðåòíîé ìîäåëè è çàïèñûâàåò ðåçóëüòàò â x ---
// ---------------------------------------------------------------------------------

__device__ __host__ void calculateDiscreteModel(double* x, const double* a, const double h)
{
	/**
	 * here we abstract from the concept of parameter names.
	 * ALL parameters are numbered with indices.
	 * In the current example, the parameters go like this:
	 * 
	 * values[0] - sym
	 * values[1] - A
	 * values[2] - B
	 * values[3] - C
	 */

	double h1 = h * a[0];
	double h2 = h * (1 - a[0]);
	x[0] = x[0] + h1 * (-x[1] - x[2]);
	x[1] = (x[1] + h1 * (x[0])) / (1 - a[1] * h1);
	x[2] = (x[2] + h1 * a[2]) / (1 - h1 * (x[0] - a[3]));
	x[2] = x[2] + h2 * (a[2] + x[2] * (x[0] - a[3]));
	x[1] = x[1] + h2 * (x[0] + a[1] * x[1]);
	x[0] = x[0] + h2 * (-x[1] - x[2]);

}


// -----------------------------------------------------------------------------------------------------
// --- Âû÷èñëÿåò òðàåêòîðèþ äëÿ îäíîé ñèñòåìû è çàïèñûâàåò ðåçóëüòàò â "data" (åñëè data != nullptr) ---
// -----------------------------------------------------------------------------------------------------

__device__ __host__ bool loopCalculateDiscreteModel(double* x, const double* values, 
	const double h, const int amountOfIterations, const int amountOfX, const int preScaller,
	int writableVar, const double maxValue, double* data, 
	const int startDataIndex, const int writeStep)
{
	double* xPrev = new double[amountOfX];
	// --- Ãëîáàëüíûé öèêë, êîòîðûé ïðîèçâîäèò âû÷èñëåíèÿ çàäàííûå amountOfIterations ðàç ---
	for ( int i = 0; i < amountOfIterations; ++i )
	{
		for (int j = 0; j < amountOfX; ++j)
		{
			xPrev[j] = x[j];
		}
		// --- Åñëè âñå-òàêè ïåðåäàëè ìàññèâ äëÿ çàïèñè - çàïèñûâàåì çíà÷åíèå ïåðåìåííîé ---
		if ( data != nullptr )
			data[startDataIndex + i * writeStep] = x[writableVar];

		// --- Ìîäåëèðóåì ñèñòåìó preScaller ðàç ( òî åñòü åñëè preScaller > 1, òî ìû ïðîïóñòèì ( preScaller - 1 ) â ñìîäåëèðîâàííîé òðàåêòîðèè ) ---
		for ( int j = 0; j < preScaller; ++j )
			calculateDiscreteModel(x, values, h);

		// --- Åñëè isnan èëè isinf - âîçâðàùàåì false, èáî ýòî íåæåëàòåëüíîå ïîâåäåíèå ñèñòåìû ---
		if ( isnan( x[writableVar] ) || isinf( x[writableVar] ) )
		{
			delete[] xPrev;
			return false;
		}

		// --- Åñëè maxValue == 0, ýòî çíà÷èò ïîëüçîâàòåëü íå âûñòàâèë îãðàíè÷åíèå, èíà÷å òðåáóåòñÿ åãî ïðîâåðèòü ---
		if ( maxValue != 0 )
			if ( fabsf( x[writableVar] ) > maxValue )
			{
				delete[] xPrev;
				return false;
			}
	}

	// --- Ïðîâåðêà íà ñâàëèâàíèå â òî÷êó ---
	double tempResult = 0;
	for (int j = 0; j < amountOfX; ++j)
	{
		tempResult += ((x[j] - xPrev[j]) * (x[j] - xPrev[j]));
	}

	if (tempResult == 0)
	{
		delete[] xPrev;
		return false;
	}

	if (sqrt(tempResult) < 1e-12)
	{
		delete[] xPrev;
		return false;
	}

	delete[] xPrev;
	return true;
}



__global__ void distributedCalculateDiscreteModelCUDA(
	const int		amountOfPointsForSkip,
	const int		amountOfThreads,
	const double	h,
	const double	hSpecial,
	double*			initialConditions,
	const int		amountOfInitialConditions,
	const double*	values,
	const int		amountOfValues,
	const int		amountOfIterations,
	const int		writableVar,
	double*			data)
{
	extern __shared__ double s[];
	double* localX = s + (threadIdx.x * amountOfInitialConditions);
	double* localValues = s + (blockDim.x * amountOfInitialConditions) + (threadIdx.x * amountOfValues);

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= amountOfThreads)
		return;

	for (int i = 0; i < amountOfInitialConditions; ++i)
		localX[i] = initialConditions[i];

	for (int i = 0; i < amountOfValues; ++i)
		localValues[i] = values[i];

	// --- Ïðîãîíÿåì ñèñòåìó amountOfPointsForSkip ðàç ( äëÿ îòðàáîòêè transientTime ) --- 
	loopCalculateDiscreteModel(localX, localValues, h, amountOfPointsForSkip,
		amountOfInitialConditions, 1, 0, 0, nullptr, 0);

	loopCalculateDiscreteModel(localX, localValues, h, idx,
		amountOfInitialConditions, 1, 0, 0, nullptr, 0, 0);

	loopCalculateDiscreteModel(localX, localValues, hSpecial, amountOfIterations,
		amountOfInitialConditions, 1, writableVar, 0, data, idx, amountOfThreads);

	return;
}



// --------------------------------------------------------------------------
// --- Ãëîáàëüíàÿ ôóíêöèÿ, êîòîðàÿ âû÷èñëÿåò òðàåêòîðèþ íåñêîëüêèõ ñèñòåì ---
// --------------------------------------------------------------------------

__global__ void calculateDiscreteModelCUDA(
	const int		nPts, 
	const int		nPtsLimiter, 
	const int		sizeOfBlock, 
	const int		amountOfCalculatedPoints, 
	const int		amountOfPointsForSkip,
	const int		dimension, 
	double*			ranges, 
	const double	h,
	int*			indicesOfMutVars, 
	double*			initialConditions,
	const int		amountOfInitialConditions, 
	const double*	values, 
	const int		amountOfValues,
	const int		amountOfIterations, 
	const int		preScaller,
	const int		writableVar, 
	const double	maxValue, 
	double*			data, 
	int*			maxValueCheckerArray)
{
	// --- Îáùàÿ ïàìÿòü â ðàìêàõ îäíîãî áëîêà ---
	// --- Ñòðîåíèå ïàìÿòè: ---
	// --- {localX_0, localX_1, localX_2, ..., localValues_0, localValues_1, ..., ñëåäóþøèé ïîòîê...} ---
	extern __shared__ double s[];

	// --- Â êàæäîì ïîòîêå ñîçäàåì óêàçàòåëü íà ïàðàìåòðû è ïåðåìåííûå, ÷òîáû ðàáîòàòü ñ íèìè êàê ñ ìàññèâàìè ---
	double* localX = s + ( threadIdx.x * amountOfInitialConditions );
	double* localValues = s + ( blockDim.x * amountOfInitialConditions ) + ( threadIdx.x * amountOfValues );

	// --- Âû÷èñëÿåì èíäåêñ ïîòîêà, â êîòîðîì íàõîäèìñÿ â äàíûé ìîìåíò ---
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= nPtsLimiter)		// Åñëè ñóùåñòâóåò ïîòîê ñ áîëüøèì èíäåêñîì, ÷åì òðåáóåòñÿ - ñðàçó çàâåðøàåì åãî
		return;

	// --- Îïðåäåëÿåì localX[] íà÷àëüíûìè óñëîâèÿìè ---
	for ( int i = 0; i < amountOfInitialConditions; ++i )
		localX[i] = initialConditions[i];

	// --- Îïðåäåëÿåì localValues[] íà÷àëüíûìè ïàðàìåòðàìè ---
	for (int i = 0; i < amountOfValues; ++i)
		localValues[i] = values[i];

	// --- Ìåíÿåì çíà÷åíèå èçìåíÿåìûõ ïàðàìåòðîâ íà ðåçóëüòàò ôóíêöèè getValueByIdx ---
	for (int i = 0; i < dimension; ++i)
		localValues[indicesOfMutVars[i]] = getValueByIdx(amountOfCalculatedPoints + idx, 
			nPts, ranges[i * 2], ranges[i * 2 + 1], i);

	// --- Ïðîãîíÿåì ñèñòåìó amountOfPointsForSkip ðàç ( äëÿ îòðàáîòêè transientTime ) --- 
	loopCalculateDiscreteModel(localX, localValues, h, amountOfPointsForSkip,
		1, amountOfInitialConditions, 0, 0, nullptr, idx * sizeOfBlock);

	// --- Òåïåðü óæå ïî-âçðîñëîìó ìîäåëèðóåì ñèñòåìó --- 
	bool flag = loopCalculateDiscreteModel(localX, localValues, h, amountOfIterations,
		amountOfInitialConditions, preScaller, writableVar, maxValue, data, idx * sizeOfBlock);

	// --- Åñëè ôóíêöèÿ ìîäåëèðîâàíèÿ âûäàëà false - çíà÷èò ìû äàæå íå áóäåì ñìîòðåòü íà ýòó ñèñòåìó â äàëüíåéøåì àíàëèçå ---
	if (!flag && maxValueCheckerArray != nullptr)
		maxValueCheckerArray[idx] = -1;	

	return;
}



__global__ void calculateDiscreteModelCUDA_H(
	const int		nPts,
	const int		nPtsLimiter,
	const int		sizeOfBlock,
	const int		amountOfCalculatedPoints,
	const double	transientTime,
	const int		dimension,
	double*			ranges,
	double*			initialConditions,
	const int		amountOfInitialConditions,
	const double*	values,
	const int		amountOfValues,
	const double	tMax,
	const int		preScaller,
	const int		writableVar,
	const double	maxValue,
	double*			data,
	int*			maxValueCheckerArray)
{
	// --- Îáùàÿ ïàìÿòü â ðàìêàõ îäíîãî áëîêà ---
	// --- Ñòðîåíèå ïàìÿòè: ---
	// --- {localX_0, localX_1, localX_2, ..., localValues_0, localValues_1, ..., ñëåäóþøèé ïîòîê...} ---
	extern __shared__ double s[];

	// --- Â êàæäîì ïîòîêå ñîçäàåì óêàçàòåëü íà ïàðàìåòðû è ïåðåìåííûå, ÷òîáû ðàáîòàòü ñ íèìè êàê ñ ìàññèâàìè ---
	double* localX = s + (threadIdx.x * amountOfInitialConditions);
	double* localValues = s + (blockDim.x * amountOfInitialConditions) + (threadIdx.x * amountOfValues);

	// --- Âû÷èñëÿåì èíäåêñ ïîòîêà, â êîòîðîì íàõîäèìñÿ â äàíûé ìîìåíò ---
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= nPtsLimiter)		// Åñëè ñóùåñòâóåò ïîòîê ñ áîëüøèì èíäåêñîì, ÷åì òðåáóåòñÿ - ñðàçó çàâåðøàåì åãî
		return;

	// --- Îïðåäåëÿåì localX[] íà÷àëüíûìè óñëîâèÿìè ---
	for (int i = 0; i < amountOfInitialConditions; ++i)
		localX[i] = initialConditions[i];

	// --- Îïðåäåëÿåì localValues[] íà÷àëüíûìè ïàðàìåòðàìè ---
	for (int i = 0; i < amountOfValues; ++i)
		localValues[i] = values[i];

	//// --- Ìåíÿåì çíà÷åíèå èçìåíÿåìûõ ïàðàìåòðîâ íà ðåçóëüòàò ôóíêöèè getValueByIdx ---
	//for (int i = 0; i < dimension; ++i)
	//	localValues[indicesOfMutVars[i]] = getValueByIdx(amountOfCalculatedPoints + idx,
	//		nPts, ranges[i * 2], ranges[i * 2 + 1], i);

	double h = pow(10, getValueByIdxLog(amountOfCalculatedPoints + idx, nPts, ranges[0], ranges[1], 0));

	// --- Ïðîãîíÿåì ñèñòåìó amountOfPointsForSkip ðàç ( äëÿ îòðàáîòêè transientTime ) --- 
	loopCalculateDiscreteModel(localX, localValues, h, transientTime / h,
		amountOfInitialConditions, 1, 0, 0, nullptr, idx * sizeOfBlock);

	// --- Òåïåðü óæå ïî-âçðîñëîìó ìîäåëèðóåì ñèñòåìó --- 
	bool flag = loopCalculateDiscreteModel(localX, localValues, h, tMax / h / preScaller,
		amountOfInitialConditions, preScaller, writableVar, maxValue, data, idx * sizeOfBlock);

	// --- Åñëè ôóíêöèÿ ìîäåëèðîâàíèÿ âûäàëà false - çíà÷èò ìû äàæå íå áóäåì ñìîòðåòü íà ýòó ñèñòåìó â äàëüíåéøåì àíàëèçå ---
	if (!flag && maxValueCheckerArray != nullptr)
		maxValueCheckerArray[idx] = -1;
	else
		maxValueCheckerArray[idx] = tMax / h / preScaller;

	return;
}



__global__ void calculateDiscreteModelICCUDA(
	const int		nPts, 
	const int		nPtsLimiter, 
	const int		sizeOfBlock, 
	const int		amountOfCalculatedPoints, 
	const int		amountOfPointsForSkip,
	const int		dimension, 
	double*			ranges, 
	const double	h,
	int*			indicesOfMutVars, 
	double*			initialConditions,
	const int		amountOfInitialConditions, 
	const double*	values, 
	const int		amountOfValues,
	const int		amountOfIterations, 
	const int		preScaller,
	const int		writableVar, 
	const double	maxValue, 
	double*			data, 
	int*			maxValueCheckerArray)
{
	// --- Îáùàÿ ïàìÿòü â ðàìêàõ îäíîãî áëîêà ---
	// --- Ñòðîåíèå ïàìÿòè: ---
	// --- {localX_0, localX_1, localX_2, ..., localValues_0, localValues_1, ..., ñëåäóþøèé ïîòîê...} ---
	extern __shared__ double s[];

	// --- Â êàæäîì ïîòîêå ñîçäàåì óêàçàòåëü íà ïàðàìåòðû è ïåðåìåííûå, ÷òîáû ðàáîòàòü ñ íèìè êàê ñ ìàññèâàìè ---
	double* localX = s + ( threadIdx.x * amountOfInitialConditions );
	double* localValues = s + ( blockDim.x * amountOfInitialConditions ) + ( threadIdx.x * amountOfValues );

	// --- Âû÷èñëÿåì èíäåêñ ïîòîêà, â êîòîðîì íàõîäèìñÿ â äàíûé ìîìåíò ---
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= nPtsLimiter)		// Åñëè ñóùåñòâóåò ïîòîê ñ áîëüøèì èíäåêñîì, ÷åì òðåáóåòñÿ - ñðàçó çàâåðøàåì åãî
		return;

	// --- Îïðåäåëÿåì localX[] íà÷àëüíûìè óñëîâèÿìè ---
	for ( int i = 0; i < amountOfInitialConditions; ++i )
		localX[i] = initialConditions[i];

	// --- Îïðåäåëÿåì localValues[] íà÷àëüíûìè ïàðàìåòðàìè ---
	for (int i = 0; i < amountOfValues; ++i)
		localValues[i] = values[i];

	// --- Ìåíÿåì çíà÷åíèå èçìåíÿåìûõ ïàðàìåòðîâ íà ðåçóëüòàò ôóíêöèè getValueByIdx ---
	for (int i = 0; i < dimension; ++i)
		localX[indicesOfMutVars[i]] = getValueByIdx( amountOfCalculatedPoints + idx, 
			nPts, ranges[i * 2], ranges[i * 2 + 1], i );

	// --- Ïðîãîíÿåì ñèñòåìó amountOfPointsForSkip ðàç ( äëÿ îòðàáîòêè transientTime ) --- 
	loopCalculateDiscreteModel(localX, localValues, h, amountOfPointsForSkip,
		amountOfInitialConditions, 1, 0, 0, nullptr, idx * sizeOfBlock);

	// --- Òåïåðü óæå ïî-âçðîñëîìó ìîäåëèðóåì ñèñòåìó --- 
	bool flag = loopCalculateDiscreteModel(localX, localValues, h, amountOfIterations,
		amountOfInitialConditions, preScaller, writableVar, maxValue, data, idx * sizeOfBlock);

	// --- Åñëè ôóíêöèÿ ìîäåëèðîâàíèÿ âûäàëà false - çíà÷èò ìû äàæå íå áóäåì ñìîòðåòü íà ýòó ñèñòåìó â äàëüíåéøåì àíàëèçå ---
	if (!flag && maxValueCheckerArray != nullptr)
		maxValueCheckerArray[idx] = -1;	

	return;
}


// --- Ôóíêöèÿ, êîòîðàÿ íàõîäèò èíäåêñ â ïîñëåäîâàòåëüíîñòè çíà÷åíèé ---
__device__ __host__ double getValueByIdx(const int idx, const int nPts,
	const double startRange, const double finishRange, const int valueNumber)
{
	return startRange + ( ( ( int )( ( int )idx / powf( ( double )nPts, ( double )valueNumber) ) % nPts )
		* ( ( double )( finishRange - startRange ) / ( double )( nPts - 1 ) ) );
}



// --- Ôóíêöèÿ, êîòîðàÿ íàõîäèò èíäåêñ â ïîñëåäîâàòåëüíîñòè çíà÷åíèé ---
__device__ __host__ double getValueByIdxLog(const int idx, const int nPts,
	const double startRange, const double finishRange, const int valueNumber)
{
	return log10(startRange) + (((int)((int)idx / powf((double)nPts, (double)valueNumber)) % nPts)
		* ((double)(log10(finishRange) - log10(startRange)) / (double)(nPts - 1)));
}



// ---------------------------------------------------------------------------------------------------
// --- Íàõîäèò ïèêè â èíòåðâàëå [startDataIndex; startDataIndex + amountOfPoints] â "data" ìàññèâå ---
// ---------------------------------------------------------------------------------------------------

__device__ __host__ int peakFinder(double* data, const int startDataIndex, 
	const int amountOfPoints, double* outPeaks, double* timeOfPeaks, double h)
{
	// --- Ïåðåìåííàÿ äëÿ õðàíåíèÿ íàéäåííûõ ïèêîâ ---
	int amountOfPeaks = 0;

	// --- Íà÷èíàåì ïðîñìàòðèâàòü çàäàííûõ èíòåðâàë íà íàëè÷èå ïèêîâ ---
	for ( int i = startDataIndex + 1; i < startDataIndex + amountOfPoints - 1; ++i )
	{
		// --- Åñëè òåêóùàÿ òî÷êà áîëüøå ïðåäûäóùåé è áîëüøå ÈËÈ ÐÀÂÍÀ ñëåäóþùåé, òî... ( íå ôàêò, ÷òî ýòî ïèê ( íàïðèìåð: 2 3 3 4 ) ) ---
		if ( data[i] > data[i - 1] && data[i] >= data[i + 1] )
		{
			// --- Îò íàéäåííîé òî÷êè íà÷èíàåì èäòè âïåðåä, ïîêà íå íàòêíåìñÿ íà òî÷êó ñòðîãî áîëüøå èëè ìåíüøå ---
			for ( int j = i; j < startDataIndex + amountOfPoints - 1; ++j )
			{
				// --- Åñëè íàòêíóëèñü íà òî÷êó ñòðîãî áîëüøå, çíà÷èò ýòî áûë íå ïèê ---
				if ( data[j] < data[j + 1] )
				{
					i = j + 1;	// --- Îáíîâëÿåì âíåøíèé ñ÷åò÷èê, ÷òîáû äâàæäû íå ïðîõîäèòü îäèí è òîò æå èíòåðâàë
					break;		// --- Âîçâðàùàåìñÿ ê âíåøíåìó öèêëó
				}
				// --- Åñëè î ÷óäî, ìû íàøëè òî÷êó ìåíüøå, ÷åì òåêóùàÿ, çíà÷èò ìû íàøëè ïèê ---
				if ( data[j] > data[j + 1] )
				{
					// --- Åñëè ìàññèê outPeaks íå ïóñò, òî äåëàåì çàïèñü ---
					if ( outPeaks != nullptr )
						outPeaks[startDataIndex + amountOfPeaks] = data[j];
					// --- Åñëè ìàññèê timeOfPeaks íå ïóñò, òî äåëàåì çàïèñü ---
					if ( timeOfPeaks != nullptr )
						timeOfPeaks[startDataIndex + amountOfPeaks] = trunc( ( (double)j + (double)i ) / (double)2 );	// Âûáèðàåì èíäåêñ ïîñåðåäèíå ìåæäó j è i
					++amountOfPeaks;
					i = j + 1; // Ïîòîìó ÷òî ñëåäóþùàÿ òî÷êà òî÷íî íå ìîæåò áûòü ïèêîì ( äâà ïèêà íå ìîãóò èäòè ïîäðÿä )
					break;
				}
			}
		}
	}
	// --- Âû÷èñëÿåì ìåæïèêîâûå èíòåðâàëû ---
	if ( amountOfPeaks > 1 ) {
		// --- Ïðîáåãàåìñÿ ïî âñåì íàéäåííûì ïèêàì è èõ èíäåêñàì ---
		for ( size_t i = 0; i < amountOfPeaks - 1; i++ )
		{
			// --- Ñìåùàåì âñå ïèêè íà îäèí èíäåêñ âëåâî, à ïåðâûé ïèê óäàëÿåì ---
			if ( outPeaks != nullptr )
				outPeaks[startDataIndex + i] = outPeaks[startDataIndex + i + 1];
			// --- Âû÷èñëÿåì ìåæïèêîâûé èíòåðâàë. Ýòî ðàçíèöà èíäåêñà ñëåäóþùåãî ïðèêà è ïðåäûäóùåãî, óìíîæåííàÿ íà øàã ---
			if ( timeOfPeaks != nullptr )
				timeOfPeaks[startDataIndex + i] = ( double )( ( timeOfPeaks[startDataIndex + i + 1] - timeOfPeaks[startDataIndex + i] ) * h );
		}
		// --- Òàê êàê îäèí ïèê óäàëèëè - âû÷èòàåì åäèíèöó èç ðåçóëüòàòà ---
		amountOfPeaks = amountOfPeaks - 1;
	}
	else {
		amountOfPeaks = 0;
	}


	return amountOfPeaks;
}



// ----------------------------------------------------------------
// --- Íàõîæäåíèå ïèêîâ â "data" ìàññèâå â ìíîãîïîòî÷íîì ðåæèìå ---
// ----------------------------------------------------------------

__global__ void peakFinderCUDA(double* data, const int sizeOfBlock, const int amountOfBlocks, 
	int* amountOfPeaks, double* outPeaks, double* timeOfPeaks, double h)
{
	// --- Âû÷èñëÿåì èíäåêñ ïîòîêà, â êîòîðîì íàõîäèìñÿ â äàíûé ìîìåíò ---
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if ( idx >= amountOfBlocks )		// Åñëè ñóùåñòâóåò ïîòîê ñ áîëüøèì èíäåêñîì, ÷åì òðåáóåòñÿ - ñðàçó çàâåðøàåì åãî
		return;

	// --- Åñëè íà ïðåäûäóùèõ ýòàïàõ ñèñòåìó óæå îòìåòèëè êàê "íåïðèãîäíóþ", òî ïðîïóñêàåì åå ---
	if ( amountOfPeaks[idx] == -1 )
	{
		amountOfPeaks[idx] = 0;
		return;
	}

	
	amountOfPeaks[idx] = peakFinder( data, idx * sizeOfBlock, sizeOfBlock, outPeaks, timeOfPeaks, h );
	return;
}



__global__ void peakFinderCUDA_H(double* data, const int sizeOfBlock, const int amountOfBlocks,
	int* amountOfPeaks, double* outPeaks, double* timeOfPeaks, double h)
{
	// --- Âû÷èñëÿåì èíäåêñ ïîòîêà, â êîòîðîì íàõîäèìñÿ â äàíûé ìîìåíò ---
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= amountOfBlocks)		// Åñëè ñóùåñòâóåò ïîòîê ñ áîëüøèì èíäåêñîì, ÷åì òðåáóåòñÿ - ñðàçó çàâåðøàåì åãî
		return;

	// --- Åñëè íà ïðåäûäóùèõ ýòàïàõ ñèñòåìó óæå îòìåòèëè êàê "íåïðèãîäíóþ", òî ïðîïóñêàåì åå ---
	if (amountOfPeaks[idx] == -1)
	{
		amountOfPeaks[idx] = 0;
		return;
	}

	amountOfPeaks[idx] = peakFinder(data, idx * sizeOfBlock, amountOfPeaks[idx], outPeaks, timeOfPeaks, h);
	return;
}



__global__ void peakFinderCUDAForCalculationOfPeriodicityByOstrovsky(double* data, const int sizeOfBlock, const int amountOfBlocks,
	int* amountOfPeaks, double* outPeaks, double* timeOfPeaks, bool* flags, double ostrovskyThreshold)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= amountOfBlocks)
		return;

	if (amountOfPeaks[idx] == -1)
	{
		amountOfPeaks[idx] = 0;
		flags[idx * 5 + 3] = true;
		return;
	}

	double lastPoint = data[idx * sizeOfBlock + sizeOfBlock - 1];

	amountOfPeaks[idx] = peakFinder(data, idx * sizeOfBlock, sizeOfBlock, outPeaks, timeOfPeaks);

	//FIRST CONDITION
	flags[idx * 5 + 0] = true;
	for (int i = idx * sizeOfBlock + 1; i < idx * sizeOfBlock + amountOfPeaks[idx]; ++i)
	{
		if (outPeaks[i] - outPeaks[i - 1] > 0)
		{
			flags[idx * 5 + 0] = false;
			break;
		}
	}

	//SECOND & THIRD CONDITION
	bool flagOne = false;
	bool flagZero = false;
	for (int i = idx * sizeOfBlock + 1; i < idx * sizeOfBlock + amountOfPeaks[idx]; ++i)
	{
		if (outPeaks[i] > ostrovskyThreshold)
			flagOne = true;
		else
			flagZero = true;
		if (flagOne && flagZero)
			break;
	}

	if (flagOne && flagZero)
		flags[idx * 5 + 1] = true;
	else
		flags[idx * 5 + 1] = false;

	if (flagOne && !flagZero)
		flags[idx * 5 + 2] = false;
	else
		flags[idx * 5 + 2] = true;

	//FOUR CONDITION
	if (amountOfPeaks[idx] == 0 || amountOfPeaks[idx] == 1)
		flags[idx * 5 + 3] = true;
	else
		flags[idx * 5 + 3] = false;

	//FIVE CONDITION
	if (lastPoint > ostrovskyThreshold)
		flags[idx * 5 + 4] = true;
	else
		flags[idx * 5 + 4] = false;
	return;
}



__device__ __host__ int kde(double* data, const int startDataIndex, const int amountOfPoints,
	int maxAmountOfPeaks, int kdeSampling, double kdeSamplesInterval1,
	double kdeSamplesInterval2, double kdeSmoothH)
{
	if (amountOfPoints == 0)
		return 0;
	if (amountOfPoints == 1 || amountOfPoints == 2)
		return 1;
	if (amountOfPoints > maxAmountOfPeaks)
		return maxAmountOfPeaks;

	double k1 = kdeSampling * amountOfPoints;
	double k2 = (kdeSamplesInterval2 - kdeSamplesInterval1) / (k1 - 1);
	double delt = 0;
	double prevPrevData2 = 0;
	double prevData2 = 0;
	double data2 = 0;
	bool strangePeak = false;
	int resultKde = 0;

	for (int w = 0; w < k1 - 1; ++w)
	{
		delt = w * k2 + kdeSamplesInterval1;
		prevPrevData2 = prevData2;
		prevData2 = data2;
		data2 = 0;
		for (int m = 0; m < amountOfPoints; ++m)
		{
			double tempData = (data[startDataIndex + m] - delt) / kdeSmoothH;
			data2 += expf(-((tempData * tempData) / 2));
		}

		if (w < 2)
			continue;
		if (strangePeak)
		{
			if (prevData2 == data2)
				continue;
			else if (prevData2 < data2)
			{
				strangePeak = false;
				continue;
			}
			else if (prevData2 > data2)
			{
				strangePeak = false;
				++resultKde;
				continue;
			}
		}
		else if (prevData2 > prevPrevData2 && prevData2 > data2)
		{
			++resultKde;
			continue;
		}
		else if (prevData2 > prevPrevData2 && prevData2 == data2)
		{
			strangePeak = true;
			continue;
		}
	}
	if (prevData2 < data2)
	{
		++resultKde;
	}
	return resultKde;
}



__global__ void kdeCUDA(double* data, const int sizeOfBlock, const int amountOfBlocks,
	int* amountOfPeaks, int* kdeResult, int maxAmountOfPeaks, int kdeSampling, double kdeSamplesInterval1,
	double kdeSamplesInterval2, double kdeSmoothH)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= amountOfBlocks)
		return;

	if (amountOfPeaks[idx] == -1)
	{
		kdeResult[idx] = 0;
		return;
	}
	kdeResult[idx] = kde(data, idx * sizeOfBlock, amountOfPeaks[idx], maxAmountOfPeaks,
		kdeSampling, kdeSamplesInterval1, kdeSamplesInterval2, kdeSmoothH);
}


// ------------------------------------------------
// --- Âû÷èñëÿåò ðàññòîÿíèå ìåæäó äâóìÿ òî÷êàìè ---
// ------------------------------------------------

__device__ __host__ double distance(double x1, double y1, double x2, double y2)
{
	if (x1 == x2 && y1 == y2)
		return 0;
	double dx = x2 - x1;
	double dy = y2 - y1;

	return hypotf(dx, dy);
}



// ----------------------
// --- Ôóíêöèÿ DBSCAN ---
// ----------------------

__device__ __host__ int dbscan(double* data, double* intervals, double* helpfulArray, 
	const int startDataIndex, const int amountOfPeaks, const int sizeOfHelpfulArray,
	const int idx, const double eps, int* outData)
{
	// ------------------------------------------------------------
	// --- Åñëè ïèêîâ 0 èëè 1 - äàæå íå îáðàáàòûâàåì ýòè ñëó÷àè ---
	// ------------------------------------------------------------

	if (amountOfPeaks <= 0)
		return 0;

	if (amountOfPeaks == 1)
		return 1;

	// ------------------------------------------------------------

	int cluster = 0;
	int NumNeibor = 0;

	for (int i = startDataIndex; i < startDataIndex + sizeOfHelpfulArray; ++i) {
		helpfulArray[i] = 0;
	}

	for (int i = 0; i < amountOfPeaks; i++)
		if (NumNeibor >= 1)
		{
			i = helpfulArray[startDataIndex + amountOfPeaks + NumNeibor - 1];
			helpfulArray[startDataIndex + amountOfPeaks + NumNeibor - 1] = 0;
			NumNeibor = NumNeibor - 1;
			for (int k = 0; k < amountOfPeaks - 1; k++) {
				if (i != k && helpfulArray[startDataIndex + k] == 0) {
					if (distance(data[startDataIndex + i], intervals[startDataIndex + i], data[startDataIndex + k], intervals[startDataIndex + k]) < eps) {
						helpfulArray[startDataIndex + k] = cluster;
						helpfulArray[startDataIndex + amountOfPeaks + k] = k;
						++NumNeibor;
					}
				}
			}
		}
		else if (helpfulArray[startDataIndex + i] == 0) {
			NumNeibor = 0;
			++cluster;
			helpfulArray[startDataIndex + i] = cluster;
			for (int k = 0; k < amountOfPeaks - 1; k++) {
				if (i != k && helpfulArray[startDataIndex + k] == 0) {
					if (distance(data[startDataIndex + i], intervals[startDataIndex + i], data[startDataIndex + k], intervals[startDataIndex + k]) < eps) {
						helpfulArray[startDataIndex + k] = cluster;
						helpfulArray[startDataIndex + amountOfPeaks + k] = k;
						++NumNeibor;
					}
				}
			}
		}

	return cluster - 1;
}



// ---------------------------------
// --- Ãëîáàëüíàÿ ôóíêöèÿ DBSCAN ---
// ---------------------------------

__global__ void dbscanCUDA(double* data, const int sizeOfBlock, const int amountOfBlocks,
	const int* amountOfPeaks, double* intervals, double* helpfulArray,
	const double eps, int* outData)
{
	// --- Âû÷èñëÿåì èíäåêñ ïîòîêà, â êîòîðîì íàõîäèìñÿ â äàíûé ìîìåíò ---
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx >= amountOfBlocks)		// Åñëè ñóùåñòâóåò ïîòîê ñ áîëüøèì èíäåêñîì, ÷åì òðåáóåòñÿ - ñðàçó çàâåðøàåì åãî
		return;

	// --- Åñëè íà ïðåäûäóùèõ ýòàïàõ ñèñòåìó óæå îòìåòèëè êàê "íåïðèãîäíóþ", òî ïðîïóñêàåì åå ---
	if (amountOfPeaks[idx] == -1)
	{
		outData[idx] = 0;
		return;
	}

	// --- Ïðèìåíÿåì àëãîðèòì dbscan ê êàæäîé ñèñòåìå
	outData[idx] = dbscan(data, intervals, helpfulArray, idx * sizeOfBlock, amountOfPeaks[idx], sizeOfBlock, idx, eps, outData);
}



// --------------------
// --- ßäðî äëÿ LLE ---
// --------------------
__global__ void LLEKernelCUDA(
	const int		nPts,
	const int		nPtsLimiter,
	const double	NT,
	const double	tMax,
	const int		sizeOfBlock,
	const int		amountOfCalculatedPoints,
	const int		amountOfPointsForSkip,
	const int		dimension,
	double*			ranges,
	const double	h,
	const double	eps,
	int*			indicesOfMutVars,
	double*			initialConditions,
	const int		amountOfInitialConditions,
	const double*	values,
	const int		amountOfValues,
	const int		amountOfIterations,
	const int		preScaller,
	const int		writableVar,
	const double	maxValue,
	double*			resultArray)
{
	extern __shared__ double s[];
	double* x = s + threadIdx.x * amountOfInitialConditions;
	double* y = s + (blockDim.x + threadIdx.x) * amountOfInitialConditions;
	double* z = s + (2 * blockDim.x + threadIdx.x) * amountOfInitialConditions;
	double* localValues = s + (3 * blockDim.x * amountOfInitialConditions) + (threadIdx.x * amountOfValues);

	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	size_t amountOfNTPoints = NT / h;

	if (idx >= nPtsLimiter)
		return;

	for (int i = 0; i < amountOfInitialConditions; ++i)
		x[i] = initialConditions[i];

	for (int i = 0; i < amountOfValues; ++i)
		localValues[i] = values[i];

	for (int i = 0; i < dimension; ++i)
		localValues[indicesOfMutVars[i]] = getValueByIdx(amountOfCalculatedPoints + idx,
			nPts, ranges[i * 2], ranges[i * 2 + 1], i);

	//printf("%f %f %f %f\n", localValues[0], localValues[1], localValues[2], localValues[3]);

	double zPower = 0;
	for (int i = 0; i < amountOfInitialConditions; ++i)
	{
		z[i] = 0.5 * (sinf(idx * (i * idx + 1) + 1));	// 0.2171828 change to z[i] = rand(0, 1) - 0.5;
		zPower += z[i] * z[i];
	}

	zPower = sqrt(zPower);

	for (int i = 0; i < amountOfInitialConditions; i++)
	{
		z[i] /= zPower;
	}


	loopCalculateDiscreteModel(x, localValues, h, amountOfPointsForSkip,
		amountOfInitialConditions, 1, 0, maxValue, nullptr, idx * sizeOfBlock);

	//Calculating

	for (int i = 0; i < amountOfInitialConditions; ++i) {
		y[i] = z[i] * eps + x[i];
	}

	double result = 0;

	for (int i = 0; i < sizeOfBlock; ++i)
	{
		bool flag = loopCalculateDiscreteModel(x, localValues, h, amountOfNTPoints,
			amountOfInitialConditions, 1, 0, maxValue, nullptr, idx * sizeOfBlock);
		if (!flag) { resultArray[idx] = 0; result;/* goto Error;*/ }

		flag = loopCalculateDiscreteModel(y, localValues, h, amountOfNTPoints,
			amountOfInitialConditions, 1, 0, maxValue, nullptr, idx * sizeOfBlock);
		if (!flag) { resultArray[idx] = 0; result;/* goto Error; */ }

		double tempData = 0;

		for (int l = 0; l < amountOfInitialConditions; ++l)
			tempData += (x[l] - y[l]) * (x[l] - y[l]);
		tempData = sqrt(tempData) / eps;

		result += log(tempData);
		
		if (tempData != 0)
			tempData = (1 / tempData);

		for (int j = 0; j < amountOfInitialConditions; ++j) {
			y[j] = (double)(x[j] - ((x[j] - y[j]) * tempData));
		}
	}

	resultArray[idx] = result / tMax;
}



// -------------------------
// --- ßäðî äëÿ LLE (IC) ---
// -------------------------
__global__ void LLEKernelICCUDA(
	const int		nPts,
	const int		nPtsLimiter,
	const double	NT,
	const double	tMax,
	const int		sizeOfBlock,
	const int		amountOfCalculatedPoints,
	const int		amountOfPointsForSkip,
	const int		dimension,
	double*			ranges,
	const double	h,
	const double	eps,
	int*			indicesOfMutVars,
	double*			initialConditions,
	const int		amountOfInitialConditions,
	const double*	values,
	const int		amountOfValues,
	const int		amountOfIterations,
	const int		preScaller,
	const int		writableVar,
	const double	maxValue,
	double*			resultArray)
{
	extern __shared__ double s[];
	double* x = s + threadIdx.x * amountOfInitialConditions;
	double* y = s + (blockDim.x + threadIdx.x) * amountOfInitialConditions;
	double* z = s + (2 * blockDim.x + threadIdx.x) * amountOfInitialConditions;
	double* localValues = s + (3 * blockDim.x * amountOfInitialConditions) + (threadIdx.x * amountOfValues);

	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	size_t amountOfNTPoints = NT / h;

	if (idx >= nPtsLimiter)
		return;

	for (int i = 0; i < amountOfInitialConditions; ++i)
		x[i] = initialConditions[i];

	for (int i = 0; i < amountOfValues; ++i)
		localValues[i] = values[i];

	for (int i = 0; i < dimension; ++i)
		x[indicesOfMutVars[i]] = getValueByIdx(amountOfCalculatedPoints + idx,
			nPts, ranges[i * 2], ranges[i * 2 + 1], i);

	//printf("%f %f %f %f\n", localValues[0], localValues[1], localValues[2], localValues[3]);

	double zPower = 0;
	for (int i = 0; i < amountOfInitialConditions; ++i)
	{
		// z[i] = sinf(0.2171828 * (i + 1) + idx + (0.2171828 + i * idx)) * 0.5;
		z[i] = 0.5 * (sinf(idx * (i * idx + 1) + 1));
		zPower += z[i] * z[i];
	}

	zPower = sqrt(zPower);

	for (int i = 0; i < amountOfInitialConditions; i++)
	{
		z[i] /= zPower;
	}


	loopCalculateDiscreteModel(x, localValues, h, amountOfPointsForSkip,
		amountOfInitialConditions, 1, 0, maxValue, nullptr, idx * sizeOfBlock);

	//Calculating

	for (int i = 0; i < amountOfInitialConditions; ++i) {
		y[i] = z[i] * eps + x[i];
	}

	double result = 0;

	for (int i = 0; i < sizeOfBlock; ++i)
	{
		bool flag = loopCalculateDiscreteModel(x, localValues, h, amountOfNTPoints,
			amountOfInitialConditions, 1, 0, maxValue, nullptr, idx * sizeOfBlock);
		if (!flag) { resultArray[idx] = 0; result;/* goto Error;*/ }

		flag = loopCalculateDiscreteModel(y, localValues, h, amountOfNTPoints,
			amountOfInitialConditions, 1, 0, maxValue, nullptr, idx * sizeOfBlock);
		if (!flag) { resultArray[idx] = 0; result;/* goto Error; */ }

		double tempData = 0;

		for (int l = 0; l < amountOfInitialConditions; ++l)
			tempData += (x[l] - y[l]) * (x[l] - y[l]);
		tempData = sqrt(tempData) / eps;

		result += log(tempData);

		if (tempData != 0)
			tempData = (1 / tempData);

		for (int j = 0; j < amountOfInitialConditions; ++j) {
			y[j] = (double)(x[j] - ((x[j] - y[j]) * tempData));
		}
	}

	resultArray[idx] = result / tMax;
}



//find projection operation (ab)
__device__ __host__ void projectionOperator(double* a, double* b, double* minuend, int amountOfValues)
{
	double numerator = 0;
	double denominator = 0;
	for (int i = 0; i < amountOfValues; ++i)
	{
		numerator += a[i] * b[i];
		denominator += b[i] * b[i];
	}

	double fraction = denominator == 0 ? 0 : numerator / denominator;

	for (int i = 0; i < amountOfValues; ++i)
		minuend[i] -= fraction * b[i];
}



__device__ __host__ void gramSchmidtProcess(double* a, double* b, int amountOfVectorsAndValuesInVector, double* denominators=nullptr/*They are is equale for our task*/)
{
	for (int i = 0; i < amountOfVectorsAndValuesInVector; ++i)
	{
		for (int j = 0; j < amountOfVectorsAndValuesInVector; ++j)
			b[j + i * amountOfVectorsAndValuesInVector] = a[j + i * amountOfVectorsAndValuesInVector];

		for (int j = 0; j < i; ++j)
			projectionOperator(a + i * amountOfVectorsAndValuesInVector,
				b + j * amountOfVectorsAndValuesInVector,
				b + i * amountOfVectorsAndValuesInVector,
				amountOfVectorsAndValuesInVector);
	}

	for (int i = 0; i < amountOfVectorsAndValuesInVector; ++i)
	{
		double denominator = 0;
		for (int j = 0; j < amountOfVectorsAndValuesInVector; ++j)
			denominator += b[i * amountOfVectorsAndValuesInVector + j] * b[i * amountOfVectorsAndValuesInVector + j];
		denominator = sqrt(denominator);
		for (int j = 0; j < amountOfVectorsAndValuesInVector; ++j)
			b[i * amountOfVectorsAndValuesInVector + j] = denominator == 0 ? 0 : b[i * amountOfVectorsAndValuesInVector + j] / denominator;

		if (denominators != nullptr)
			denominators[i] = denominator;
	}
}



__global__ void LSKernelCUDA(
	const int nPts,
	const int nPtsLimiter,
	const double NT,
	const double tMax,
	const int sizeOfBlock,
	const int amountOfCalculatedPoints,
	const int amountOfPointsForSkip,
	const int dimension,
	double* ranges,
	const double h,
	const double eps,
	int* indicesOfMutVars,
	double* initialConditions,
	const int amountOfInitialConditions,
	const double* values,
	const int amountOfValues,
	const int amountOfIterations,
	const int preScaller,
	const int writableVar,
	const double maxValue,
	double* resultArray)
{
	extern __shared__ double s[];

	unsigned long long buferForMem = 0;
	double* x = s + threadIdx.x * amountOfInitialConditions;

	buferForMem += blockDim.x * amountOfInitialConditions;
	double* y = s + buferForMem + amountOfInitialConditions * amountOfInitialConditions * threadIdx.x;

	buferForMem += blockDim.x * amountOfInitialConditions * amountOfInitialConditions;
	double* z = s + buferForMem + amountOfInitialConditions * amountOfInitialConditions * threadIdx.x;

	buferForMem += blockDim.x * amountOfInitialConditions * amountOfInitialConditions;
	double* localValues = s + buferForMem + amountOfValues * threadIdx.x;

	buferForMem += blockDim.x * amountOfValues;
	double* result = s + buferForMem + amountOfInitialConditions * threadIdx.x;

	buferForMem += blockDim.x * amountOfInitialConditions;
	double* denominators = s + buferForMem + amountOfInitialConditions * threadIdx.x;

	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	size_t amountOfNTPoints = NT / h;

	if (idx >= nPtsLimiter)
		return;

	for (int i = 0; i < amountOfInitialConditions; ++i)
	{
		x[i] = initialConditions[i];
		result[i] = 0;
		denominators[i] = 0;
	}

	for (int i = 0; i < amountOfValues; ++i)
		localValues[i] = values[i];

	for (int i = 0; i < dimension; ++i)
		localValues[indicesOfMutVars[i]] = getValueByIdx(amountOfCalculatedPoints + idx,
			nPts, ranges[i * 2], ranges[i * 2 + 1], i);

	for (int j = 0; j < amountOfInitialConditions; ++j)
	{
		double zPower = 0;
		for (int i = 0; i < amountOfInitialConditions; ++i)
		{
			z[j * amountOfInitialConditions + i] = sinf(0.2171828 * (i + 1) * (j + 1) + idx + (0.2171828 + i * j * idx)) * 0.5;//0.5 * (sinf(idx * ((1 + i + j) * idx + 1) + 1));	// 0.2171828 change to z[i] = rand(0, 1) - 0.5;
			zPower += z[j * amountOfInitialConditions + i] * z[j * amountOfInitialConditions + i];
		}

		zPower = sqrt(zPower);

		for (int i = 0; i < amountOfInitialConditions; i++)
		{
			z[j * amountOfInitialConditions + i] /= zPower;
		}
	}


	loopCalculateDiscreteModel(x, localValues, h, amountOfPointsForSkip,
		amountOfInitialConditions, 1, 0, maxValue, nullptr, idx * sizeOfBlock);

	//Calculating


	gramSchmidtProcess(z, y, amountOfInitialConditions);


	for (int j = 0; j < amountOfInitialConditions; ++j)
	{
		for (int i = 0; i < amountOfInitialConditions; ++i) {
			y[j * amountOfInitialConditions + i] = y[j * amountOfInitialConditions + i] * eps + x[i];
		}
	}

	//double result = 0;

	for (int i = 0; i < sizeOfBlock; ++i)
	{
		bool flag = loopCalculateDiscreteModel(x, localValues, h, amountOfNTPoints,
			amountOfInitialConditions, 1, 0, maxValue, nullptr, idx * sizeOfBlock);
		if (!flag) { for (int m = 0; m < amountOfInitialConditions; ++m ) resultArray[idx * amountOfInitialConditions + m] = 0;/* goto Error;*/ }

		for (int j = 0; j < amountOfInitialConditions; ++j)
		{
			flag = loopCalculateDiscreteModel(y + j * amountOfInitialConditions, localValues, h, amountOfNTPoints,
				amountOfInitialConditions, 1, 0, maxValue, nullptr, idx * sizeOfBlock);
			if (!flag) { for (int m = 0; m < amountOfInitialConditions; ++m) resultArray[idx * amountOfInitialConditions + m] = 0;/* goto Error; */ }
		}

		//I'M STOPPED HERE!!!!!!!!!!!!

		//__syncthreads();

		//NORMALIZTION??????????
		// 
		for (int k = 0; k < amountOfInitialConditions; ++k)
			for (int l = 0; l < amountOfInitialConditions; ++l)
				y[k * amountOfInitialConditions + l] = y[k * amountOfInitialConditions + l] - x[l];

		gramSchmidtProcess(y, z, amountOfInitialConditions, denominators);

		//denominator[amountOfInitialConditions];

		for (int k = 0; k < amountOfInitialConditions; ++k)
		{
			result[k] += log(denominators[k] / eps);

			for (int j = 0; j < amountOfInitialConditions; ++j) {
				y[k * amountOfInitialConditions + j] = (double)(x[j] + z[k * amountOfInitialConditions + j] * eps);
			}
		}
	}

	for (int i = 0; i < amountOfInitialConditions; ++i)
		resultArray[idx * amountOfInitialConditions + i] = result[i] / tMax;
}
